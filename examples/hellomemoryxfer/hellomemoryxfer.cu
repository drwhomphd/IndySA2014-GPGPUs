#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <unistd.h>

#include <hip/hip_runtime.h>

float *hellocuda_gpu_interface (float *, float *, int);
__global__ void hellocuda_kernel(float *x, float *y, float *ans, int num_floats);

int main(int argc, char **argv) {

  // We'll use the first argument of the input for the size of our floats.
  int count = atoi(argv[1]);

  printf("Count: %d\n", count);

  float x[count];
  float y[count];

  // We're going to generate count random numbers
  FILE *rng = fopen ("/dev/urandom", "r");
  fread(x, sizeof(float), count, rng);
  fread(y, sizeof(float), count, rng);
  
  // We use these for benchmark timing
  hipEvent_t gpuStart, gpuStop;
  float runtime = 0;
  hipEventCreate(&gpuStart);
  hipEventCreate(&gpuStop);

  hipEventRecord(gpuStart,0); // Start event recording
  float *res = hellocuda_gpu_interface(x, y, count);
  hipEventRecord(gpuStop,0); //  Stop event recording
  hipEventSynchronize(gpuStop);
  hipEventElapsedTime( &runtime, gpuStart, gpuStop);

  printf("Elapsed Time: %f\n", runtime);

  free(res);

  return 0;
  
}

float *hellocuda_gpu_interface (float *x, float *y, int num_floats) {

  float *x_device;
  float *y_device;

  float *ans_device;
  float *ans_host = (float *) malloc (sizeof(float) * num_floats);

  // Allocate our device memory
  hipMalloc( (void **) &x_device, num_floats * sizeof(float) );
  hipMalloc( (void **) &y_device, num_floats * sizeof(float) );
  hipMalloc( (void **) &ans_device, num_floats * sizeof(float) );

  // Transfer our buffer to the device
  hipMemcpy(x_device, x, sizeof(float) * num_floats, hipMemcpyHostToDevice);
  hipMemcpy(y_device, y, sizeof(float) * num_floats, hipMemcpyHostToDevice);

  // Calculate the block size
  int threads_per_block = 192; // 1 SMX Per Block
  int num_blocks = (num_floats + (threads_per_block - (num_floats % threads_per_block))) / threads_per_block;

  // Execute our kernel
  hellocuda_kernel<<< num_blocks, threads_per_block >>>(x_device, y_device, ans_device, num_floats);

  // Transfer our answer off the device
  hipMemcpy(ans_host, ans_device, sizeof(float) * num_floats, hipMemcpyDeviceToHost);

  return ans_host;
}

__global__ void hellocuda_kernel(float *x, float *y, float *ans, int num_floats) {

  register const uint32_t full_thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (full_thread_id < num_floats) {
    ans[full_thread_id] = x[full_thread_id] * y[full_thread_id];
  }

}

